
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void MatrixMultiplication(float* M, float* N, float* P, int height, int width, int depth)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < depth) {
        float pvalue = 0;
        for (int k = 0; k < width; k++) {
            pvalue += M[row * width + k] * N[k * depth + col];
        }
        P[row * depth + col] = pvalue;
    }
}

void print_matrix(float* matrix, int N, int M) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("%f ", matrix[i * M + j]);
        }
        printf("\n");
    }
}

int main() {
    int N = 1024; 
    int M = 512;
    int K = 2048;
    
    float *h_M = (float*)malloc(N * M * sizeof(float));
    float *h_N = (float*)malloc(M * K * sizeof(float));
    float *h_P = (float*)malloc(N * K * sizeof(float));

    srand(time(NULL));
    for (int i = 0; i < N * M; i++) {
        h_M[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < M * K; i++) {
        h_N[i] = rand() / (float)RAND_MAX;
    }

    float *d_M, *d_N, *d_P;
    hipMalloc((void**)&d_M, N * M * sizeof(float));
    hipMalloc((void**)&d_N, M * K * sizeof(float));
    hipMalloc((void**)&d_P, N * K * sizeof(float));

    hipMemcpy(d_M, h_M, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, M * K * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 16; // number of threads per block
    dim3 threadsPerBlock(block_size, block_size);
    dim3 numBlocks(ceil(K / (float)block_size), ceil(N / (float)block_size));
    
    hipEvent_t start, stop;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    MatrixMultiplication<<<numBlocks, threadsPerBlock>>>(d_M, d_N, d_P, N, M, K);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_P, d_P, N * K * sizeof(float), hipMemcpyDeviceToHost);

    printf("Elapsed time: %fsec\n", elapsed_time/100);

    free(h_M);
    free(h_N);
    free(h_P);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}
